#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "MAP-UOT_GPU.h"

int main(int argc, char **argv) {
	int m = atoi(argv[1]);
	int n = atoi(argv[2]);

	// data preprocess -- MAP-UOT
	float *uot_0 = new float[m * n];
	float *distri_row_0 = new float[m];
	float *distri_col_0 = new float[n];
	float *sum_row0_0 = new float[m];
	float *sum_col0_0 = new float[n];
	float *sum_col0_1 = new float[n];
	for(int i = 0; i < m * n; ++i) {
		uot_0[i] = (float)rand() / (float)(RAND_MAX) * 100;
	}
	for(int i = 0; i < m; ++i) {
		distri_row_0[i] = (float)rand() / (float)(RAND_MAX) * 100;
	}
	for(int i = 0; i < n; ++i) {
		distri_col_0[i] = (float)rand() / (float)(RAND_MAX) * 100;
	}
	int count_0 = 0;
	float error_0 = 1;
	float fi = 0.5;
	for(int i = 0; i < n; ++i) {
		for(int j = 0; j < m; ++j) {
			sum_col0_0[i] += uot_0[i + j * n];
		}
	}
	for(int i = 0; i < m; ++i) {
		for(int j = 0; j < n; ++j) {
			sum_row0_0[i] += uot_0[i * n + j];
		}
	}

	// data preprocess -- Typical-UOT
	float *uot_1 = new float[m * n];
	float *distri_row_1 = new float[m];
	float *distri_col_1 = new float[n];
	float *sum_col1_0 = new float[n];
	float *sum_col1_1 = new float[n];
	for(int i = 0; i < m * n; ++i) {
		uot_1[i] = uot_0[i];
	}
	for(int i = 0; i < m; ++i) {
		distri_row_1[i] = distri_row_0[i];
	}
	for(int i = 0; i < n; ++i) {
		distri_col_1[i] = distri_col_0[i];
	}

	// test Typical-UOT
	//UOT_typical(uot_1, distri_row_1, distri_col_1, sum_col1_0, sum_col1_1, m, n, fi);
	
	// test MAP-UOT
	UOT_GPUMAP(uot_0, distri_row_0, distri_col_0, sum_row0_0, sum_col0_0, sum_col0_1, error_0, m, n, fi, count_0);	
	
	// compute error
	//compute_error(uot_0, uot_1, m, n);
	
	return 0;
}

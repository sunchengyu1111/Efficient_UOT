#include "hip/hip_runtime.h"
#include "MAP-UOT_TestGPUMap.h"

void UOT_GPUMAP(float *uot_0, float *distri_row_0, float *distri_col_0, float *sum_row0_0, float *sum_col0_0, float *sum_col0_1, float error_0, const int m, const int n, const double fi, int count_0) {
	float *d_uot_0, *d_distri_row_0, *d_distri_col_0, *d_sum_row0_0, *d_sum_col0_0, *d_sum_col0_1, *d_error_0;
        hipMalloc((float**)&d_uot_0, m * n * sizeof(float));
        hipMalloc((float**)&d_distri_row_0, m * sizeof(float));
        hipMalloc((float**)&d_distri_col_0, n * sizeof(float));
        hipMalloc((float**)&d_sum_row0_0, m * sizeof(float));
        hipMalloc((float**)&d_sum_col0_0, n * sizeof(float));
        hipMalloc((float**)&d_sum_col0_1, n * sizeof(float));
        hipMalloc((float**)&d_error_0, sizeof(float));
        hipMemcpy(d_uot_0, uot_0, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_distri_row_0, distri_row_0, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_distri_col_0, distri_col_0, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sum_row0_0, sum_row0_0, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sum_col0_0, sum_col0_0, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sum_col0_1, sum_col0_1, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_error_0, &error_0, sizeof(float), hipMemcpyHostToDevice);
        // parameter
	double start = 0, finish = 0; // ,start0, finish0, start1, finish1;
        const int temp = 128;
        const int c_block_x = 128;
        const int c_iter_y = 8;
        dim3 c_block(c_block_x);
        // add '/4'
        dim3 c_grid((n + c_block_x - 1) / c_block_x, (m + c_iter_y - 1) / c_iter_y);
        const int r_block_x = 128;
        const int r_block_y = 2;
        const int r_iter_y = 8;
        dim3 r_block(r_block_x, r_block_y);
        // add '/4'
        dim3 r_grid((n + r_block_x - 1) / r_block_x, (m + r_block_y * r_iter_y - 1) / (r_block_y * r_iter_y));
        double time = 0; // ,time0 = 0, time1 = 0;
	while(error_0 >= eps) {
                //printf("count_0 = %d.\n", count_0);
                count_0++;
                start = cpuSecond();
                hipMemset(d_error_0, 0, sizeof(float));
                hipMemcpy(d_sum_col0_1, d_sum_col0_0, n * sizeof(float), hipMemcpyDeviceToDevice);
                modify_col_factor<<<(n + temp - 1) / temp, temp>>>(d_distri_col_0, d_sum_col0_0, fi);
                hipMemset(d_sum_row0_0, 0, m * sizeof(float));
                UOT_iterbase_colsum<<<c_grid, c_block>>>(d_uot_0, d_sum_row0_0, d_sum_col0_0, m, n, c_iter_y);
                modify_row_factor<<<(n + temp - 1) / temp, temp>>>(d_distri_row_0, d_sum_row0_0, fi);
                hipMemset(d_sum_col0_0, 0, n * sizeof(float));
                UOT_iterbase_rowsum<<<r_grid, r_block>>>(d_uot_0, d_sum_row0_0, d_sum_col0_0, m, n, r_iter_y);
                UOT_error<<<(n + temp - 1) / temp, temp>>>(d_sum_col0_0, d_sum_col0_1, d_error_0);
                hipDeviceSynchronize();
                finish = cpuSecond() - start;
                time += finish;
                hipMemcpy(sum_row0_0, d_sum_row0_0, m * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(sum_col0_0, d_sum_col0_0, n * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(sum_col0_1, d_sum_col0_1, n * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(uot_0, d_uot_0, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(&error_0, d_error_0, sizeof(float), hipMemcpyDeviceToHost);
                //printf("error_0 = %f.\n", error_0);               
                //printMV(uot_0, distri_row_0, distri_col_0, sum_row0_0, sum_col0_0, sum_col0_1, error_0, m, n);
        }
        printf("%f, ", time / count_0);
}

__global__ void modify_row_factor(float *distri_row, float *sum_row, float fi) {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        sum_row[id] = powf(distri_row[id] / sum_row[id], fi);
}

__global__ void modify_col_factor(float *distri_col, float *sum_col, float fi) {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        sum_col[id] = powf(distri_col[id] / sum_col[id], fi);
}

__global__ void UOT_error(float *sum_col0, float *sum_col1, float *error) {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        int laneID = threadIdx.x % 32;
        float t = abs(sum_col1[id] - sum_col0[id]);
        for(unsigned w = 16; w >= 1; w >>= 1) {
                t += __shfl_down_sync(0xffffffff, t, w);
        }
        if(laneID == 0) atomicAdd(error, t);
}

__global__ void UOT_iterbase_rowsum(float *uot, float *sum_row, float *sum_col, int m, int n, int iter_y) {
        // -----change along with blockDim.y----- //
        __shared__ float sm_col[2][128];
        __shared__ float sm_row[16];
        int col = blockDim.x * blockIdx.x + threadIdx.x;
        int row = iter_y * (blockDim.y * blockIdx.y + threadIdx.y);
        int idx = row * n + col;
        sm_col[threadIdx.y][threadIdx.x] = 0;
        // -----change along with iter_y, blockDim.y----- //
        if(threadIdx.x < 16 && threadIdx.y == 0) sm_row[threadIdx.x] = sum_row[threadIdx.x + row];
        __syncthreads();
        int sm_rowid = iter_y * threadIdx.y;
        #pragma unroll
        for(int i = 0; i < iter_y; ++i) {
                float t = uot[idx];
                t *= sm_row[sm_rowid];
                sm_col[threadIdx.y][threadIdx.x] += t;
                uot[idx] = t;
                sm_rowid++;
                idx += n;
        }
        __syncthreads();
        // change along with blockDim.y
        if(threadIdx.y == 0) atomicAdd(&sum_col[col], sm_col[0][threadIdx.x] + sm_col[1][threadIdx.x]); // + sm_col[2][laneID] + sm_col[3][laneID]);
}

__global__ void UOT_iterbase_colsum(float *uot, float *sum_row, float *sum_col, int m, int n, int iter_y) {
        int col = blockDim.x * blockIdx.x + threadIdx.x;
        int row = iter_y * blockIdx.y;
        int idx = row * n + col;
        int laneID = threadIdx.x % 32;
        int offset = threadIdx.x / 32;
        // -----change along with blockDim.x----- //
        __shared__ float sm_col0[128];
        __shared__ float sm_col1[4];
        //float *p_sum_col = &sum_col[col];
        sm_col0[threadIdx.x] = sum_col[col];
        #pragma unroll
        for(int i = 0; i < iter_y; ++i) {
                float t = uot[idx];
                t *= sm_col0[threadIdx.x];
                uot[idx] = t;
                for(unsigned w = 16; w >= 1; w >>= 1) {
                        t += __shfl_down_sync(0xffffffff, t, w);
                }
                if(laneID == 0) sm_col1[offset] = t;
                __syncthreads();
                // -----change along with blockDim.x----- //
                if(threadIdx.x == 0) atomicAdd(&sum_row[row], sm_col1[0] + sm_col1[1] + sm_col1[2] + sm_col1[3]);
                //if(threadIdx.x == 1) atomicAdd(&sum_row[row], sm_col1[4] + sm_col1[5] + sm_col1[6] + sm_col1[7]);
                //if(threadIdx.x == 2) atomicAdd(&sum_row[row], sm_col1[8] + sm_col1[9] + sm_col1[10] + sm_col1[11]);
                //if(threadIdx.x == 3) atomicAdd(&sum_row[row], sm_col1[12] + sm_col1[13] + sm_col1[14] + sm_col1[15]);
                __syncthreads();
                row++;
                idx += n;
        }
}
